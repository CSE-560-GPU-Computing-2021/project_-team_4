#include "hip/hip_runtime.h"
#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION

#include <iostream>
#include <fstream>
#include <random>
#include <ctime>
#include <hip/hip_runtime.h>

#include "Utils.h"
#include "Canvas.h"
#include "Tree.h"
#include "collision_gpu.h"
#include <chrono>
#define ENABLE_TIMER 1

using namespace std;
extern "C" int build_tree(Tree *tree1, Tree *tree2){
    if(print_file){ std::ofstream myfile; myfile.open ("path_raw.txt", std::ios_base::trunc | std::ios_base::out); myfile.close();}
    std::ofstream myfile;
    myfile.open ("path_raw.txt", std::ios_base::app);
    std::default_random_engine generator;
    generator.seed(SEED);
    float time_gpu = 0.;

    std::uniform_real_distribution<> sample_dir_x(BB.first.first, BB.second.first);
    std::uniform_real_distribution<> sample_dir_y(BB.first.second, BB.second.second);
    int iterations = 0;
    Tree* tree = tree1;
    Tree* sec_tree = tree2;

    bool* collided;
    bool* col_host;
    hipMalloc((void **)&collided, sizeof(bool));
    col_host = (bool*)malloc(sizeof(bool));
    while (iterations++<MAX_ITERATIONS){
        if(iterations%2==0){tree = tree1; sec_tree = tree2;} else{tree = tree2; sec_tree = tree1;}
        Node sampled_dir = std::make_pair(sample_dir_x(generator), sample_dir_y(generator));
        Node parent = Utils::nearest_neighbour(sampled_dir, tree->nodes);
        Node new_node = Utils::extend(parent, sampled_dir, STEP_SIZE);

        std::chrono::high_resolution_clock::time_point t1 = std::chrono::high_resolution_clock::now();


        hipMemset(collided, false, sizeof(bool));
        is_colliding_gpu(new_node, tree->canvas->obstacles_gpu, collided, tree->canvas->obstacles.size());


        hipMemcpy(col_host, collided, sizeof (bool), hipMemcpyDeviceToHost);

        if(SINGLE_CHECK_TIME) {
//            std::cout << *col_host << "\n";
            std::chrono::high_resolution_clock::time_point t2 = std::chrono::high_resolution_clock::now();
            std::chrono::duration<double, std::milli> time_span = t2 - t1;
            time_gpu += time_span.count();
        }

        if (tree->add_node(parent, new_node, *col_host)){
            if(print_file) {
                myfile << parent.first << " " << parent.second << " " <<  new_node.first << " " << new_node.second << " " << tree->id << "\n";
            }
            if (Utils::eul_dist(new_node, tree->end)<0.2){
                myfile.close();
                return iterations;
            }
            for(auto node: sec_tree->nodes){
                if (Utils::eul_dist(new_node, node)<0.2){
                    myfile.close();
                    if(SINGLE_CHECK_TIME){std::cout << "Time GPU: " << time_gpu << " ms" << std::endl;}
                    return iterations;
                }
            }
        }
        if(PRINT_ITERATIONS){if(iterations%1000==0){std::cout<< iterations << std::endl;}}
    }
    myfile.close();
    return -1;
}

int main(int argc, char **argv) {
    Canvas canvas;
    canvas.add_obs_from_file(path_to_obs);

    Node obs_array[canvas.obstacles.size()];
    std::copy(canvas.obstacles.begin(), canvas.obstacles.end(), obs_array);
    hipMalloc((void **)&canvas.obstacles_gpu, canvas.obstacles.size()*sizeof(Node));
    hipMemcpy(canvas.obstacles_gpu, obs_array, canvas.obstacles.size()*sizeof(Node), hipMemcpyHostToDevice);

    Tree tree1(&canvas, start_node, end_node, STEP_SIZE, 0);
    Tree tree2(&canvas, end_node, start_node, STEP_SIZE, 1);

    std::chrono::high_resolution_clock::time_point t1 = std::chrono::high_resolution_clock::now();
    int iterations = build_tree(&tree1, &tree2);
    std::chrono::high_resolution_clock::time_point t2 = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> time_span = t2 - t1;
    std::cout << "Time spent: " << time_span.count() << " ms" << std::endl;
    std::cout << iterations << std::endl;
    return 0;
}
