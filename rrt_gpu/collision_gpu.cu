#include "hip/hip_runtime.h"
#include "config.h"
#include "Utils.h"
#include "collision_gpu.h"
#include "iostream"
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

__device__ float eul_dist(std::pair<float, float> node1, std::pair<float, float> node2) {
    return sqrt(pow(node1.first - node2.first, 2) +  pow(node1.second - node2.second, 2));
}
__global__ void collision(Node node, Node* obstacles, bool* col, int obs_size){
    int obs_ind = blockDim.x*blockIdx.x + threadIdx.x;
    if(obs_ind < obs_size) {
        if (eul_dist(node, obstacles[obs_ind]) < clearance) {
//            printf("%d \n", obs_ind);
            *col = true;
        }
    }
}

void is_colliding_gpu(Node node, Node* obstacles, bool* col, int obs_size) {
//    std::cout << "Size: " <<obs_size <<"\n";/
    collision<<<(obs_size-1)/1024+1, 1024>>>(node, obstacles, col, obs_size);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
    hipDeviceSynchronize();
//    std::cout << "check: " << *col << "\n";
}

